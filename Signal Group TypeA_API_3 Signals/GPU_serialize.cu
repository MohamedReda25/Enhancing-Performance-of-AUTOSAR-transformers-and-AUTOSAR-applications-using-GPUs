#include "hip/hip_runtime.h"
#include "GPU_headers.h"
#include <hip/hip_runtime.h>
using namespace std;


__global__ void ComXf_Com_ComSignalGroupA_k(uint8* buffer, uint32* bufferLength, SignalGroup_A_Type* dataElement, uint32* offsets1) {
    uint8* buff = buffer;
    uint32 x = blockIdx.x * blockDim.x + threadIdx.x;
    SignalGroup_A_Type* dataElement_ptr = dataElement;
    if (x < NUM_BYTES) {
        uint32 idx = x;
        if (idx >= offsets1[0] && idx < offsets1[1]) {
            buff[idx] = *(((uint8*)&dataElement_ptr->signal1) + (idx - offsets1[0]));
        }
        if (idx >= offsets1[2] && idx < offsets1[3]) {
            buff[idx] = *(((uint8*)&dataElement_ptr->signal2) + (idx - offsets1[2]));
        }
        if (idx >= offsets1[4] && idx < offsets1[5]) {
            buff[idx] = *(((uint8*)&dataElement_ptr->signal3) + (idx - offsets1[4]));
        }
    }
    
    
}

void ComXf_Com_ComSignalGroupA(uint8* buffer, uint32* bufferLength, SignalGroup_A_Type dataElement) {

    uint32 block_size = 1024;
    uint32 grid_size = (NUM_BYTES + block_size - 1) / block_size;

    uint32* bufferlength = bufferLength;

    SignalGroup_A_Type* ptr_s = &dataElement;
    SignalGroup_A_Type* dev_ptr_s = 0;


    
    // Allocate device memory
    hipError_t cudaStatus = hipMalloc((void**)&dev_ptr_s, sizeof(SignalGroup_A_Type));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // Transfer data from host to device
    cudaStatus = hipMemcpy(dev_ptr_s, ptr_s, sizeof(SignalGroup_A_Type), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("hipMemcpy failed (HostToDevice): %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_ptr_s);  // Free the allocated device memory
    }


    //Start & End offsets of Signals 2 - 10
    const uint32 offsets_size_k1 = 6;
    uint32 offsets_k1[offsets_size_k1];
    uint32* d_offsets_k1;

    uint32 start = 0, end = sizeof(dev_ptr_s->signal1);
    offsets_k1[0] = start;
    offsets_k1[1] = end;

    start = end;
    end = start + sizeof(dev_ptr_s->signal2);
    offsets_k1[2] = start;
    offsets_k1[3] = end;

    start = end;
    end = start + sizeof(dev_ptr_s->signal3);
    offsets_k1[4] = start;
    offsets_k1[5] = end;

    






    printf("Total Number of Bytes: %d\n", end);

    hipMalloc((void**)&d_offsets_k1, offsets_size_k1 * sizeof(int));
    hipMemcpy(d_offsets_k1, offsets_k1, offsets_size_k1 * sizeof(int), hipMemcpyHostToDevice);

    //Serialization Sandwich
    //I'm here

    ComXf_Com_ComSignalGroupA_k<< <grid_size, block_size >> > ((uint8*)buffer, bufferlength, dev_ptr_s, d_offsets_k1);

    hipDeviceSynchronize();
    

    
    
    //Serialization Sandwich





    hipFree(dev_ptr_s);
    hipFree(d_offsets_k1);


}


