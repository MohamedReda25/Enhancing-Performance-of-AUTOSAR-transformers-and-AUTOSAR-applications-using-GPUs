﻿#include "GPU_headers.h"
#include "stdio.h"
#include "stdlib.h"
#include "iostream"


void fill_struct_with_data(SignalGroup_A_Type* s_ptr) {
    for (int i = 0; i < signal1_size; i++) {
        s_ptr->signal1[i] = 'a' + (i % 26);
    }
    for (int i = 0; i < signal2_size; i++) {
        s_ptr->signal2[i] = 'a' + (i % 26);
    }
    for (int i = 0; i < signal3_size; i++) {
        s_ptr->signal3[i] = 'a' + (i % 26);
    }


}



void fill_struct_with_zeros(SignalGroup_A_Type* s_ptr) {
    for (int i = 0; i < signal1_size; i++) {
        s_ptr->signal1[i] = 0;
    }
    for (int i = 0; i < signal2_size; i++) {
        s_ptr->signal2[i] = 0;
    }
    for (int i = 0; i < signal3_size; i++) {
        s_ptr->signal3[i] = 0;
    }


}




void serialization_time(float64 duration_s) {
    FILE* serialize;
    char serialize_filename[] = "Time of serialization.txt";

    serialize = fopen(serialize_filename, "w");
    fprintf(serialize, "%f", duration_s * 1e-3);
}
void deserialization_time(float64 duration_s_d) {
    FILE* deserialize;
    char deserialize_filename[] = "Time of deserialization.txt";

    deserialize = fopen(deserialize_filename, "w");


    fprintf(deserialize, "%f", duration_s_d * 1e-3);

}



 





void buffer_data_after_serialization_file_creation(uint8* buffer) {
    FILE* serialize_data;
    char serialize_data_filename[] = "Buffer after serialization";
    //fprintf(serialize_data, "%s", "Buffer data after serialization:\n");
    serialize_data = fopen(serialize_data_filename, "w");
    for (uint32 i = 0; i < buffer_length; i++) {

        fprintf(serialize_data, "%c", buffer[i]);
        fprintf(serialize_data, "%s", "\n");

    }
}

void struct_data_after_deserialization_file_creaation(SignalGroup_A_Type* d_ptr) {
    FILE* deserialize_data;
    char deserialize_data_filename[] = "Struct after deserialization";
    deserialize_data = fopen(deserialize_data_filename, "w");
    for (int i = 0; i < signal1_size; i++) {

        fprintf(deserialize_data, "%c", *((uint8*)(&(d_ptr->signal1)) + i));
        fprintf(deserialize_data, "%s", "\n");

    }
    fprintf(deserialize_data, "%s", "End of signal 1--------------------------------------\n");
    for (int i = 0; i < signal2_size; i++) {

        fprintf(deserialize_data, "%c", *((uint8*)(&(d_ptr->signal2)) + i));
        fprintf(deserialize_data, "%s", "\n");

    }
    fprintf(deserialize_data, "%s", "End of signal 2--------------------------------------\n");
    for (int i = 0; i < signal3_size; i++) {

        fprintf(deserialize_data, "%c", *((uint8*)(&(d_ptr->signal3)) + i));
        fprintf(deserialize_data, "%s", "\n");

    }
    fprintf(deserialize_data, "%s", "End of signal 3--------------------------------------\n");

}





int main()
{
    //Struct is Allocated Using CUDAMemCpy
    SignalGroup_A_Type s;
    SignalGroup_A_Type d;
    SignalGroup_A_Type* d_ptr = &d;
    SignalGroup_A_Type* s_ptr = &s;

    
    fill_struct_with_data(s_ptr);


    //Buffer is in Unified Memory Allocation

    uint32 bufflength = buffer_length;
    uint32* bufflength_ptr = &bufflength;
    uint8* buffer = nullptr;
    hipError_t cudaStatus;

    hipFree(0);
    cudaStatus = hipMallocManaged(&buffer, *(bufflength_ptr));
    if (buffer == nullptr) {
        printf("Failed to allocate memory for buffer.\n");
    }
    if (cudaStatus != hipSuccess) { 
        fprintf(stderr, "hipMallocManaged failed: %s\n", hipGetErrorString(cudaStatus));
    }

   





    // API CALL
    auto start_time = std::chrono::high_resolution_clock::now();
    ComXf_Com_ComSignalGroupA(buffer, bufflength_ptr, s);
    auto finish_time = std::chrono::high_resolution_clock::now();

    auto duration_ns = std::chrono::duration_cast<std::chrono::nanoseconds>(finish_time - start_time);  //Time in NANOSEC
    double duration_s = duration_ns.count();
    printf("\nTime Of Serialization In Micro Seconds: %f\n", duration_s * 1e-3);  //Time in MICROSEC



    fill_struct_with_zeros(d_ptr);

    


    auto start_time_d = std::chrono::high_resolution_clock::now();
    ComXf_Inv_Com_ComSignalGroupA(buffer, bufflength, d_ptr);
    auto finish_time_d = std::chrono::high_resolution_clock::now();

    auto duration_ns_d = std::chrono::duration_cast<std::chrono::nanoseconds>(finish_time_d - start_time_d);  //Time in NANOSEC
    float64 duration_s_d = duration_ns_d.count();
    printf("\nTime Of Deserialization In Micro Seconds: %f\n", duration_s_d * 1e-3);  //Time in MICROSEC






    serialization_time(duration_s);
   
    deserialization_time(duration_s_d);

    buffer_data_after_serialization_file_creation(buffer);

    struct_data_after_deserialization_file_creaation(d_ptr);

    





    
    // Prefetch to the host (CPU)
    cudaStatus = hipFree(buffer);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFree failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }
    return(0);
}
