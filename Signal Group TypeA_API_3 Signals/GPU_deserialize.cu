#include "hip/hip_runtime.h"
#include "GPU_headers.h"
#include <hip/hip_runtime.h>
using namespace std;


__global__ void ComXf_Inv_Com_ComSignalGroupA_k(uint8* buffer, uint32 bufferLength, SignalGroup_A_Type* dataElement, uint32* offsets1) {

    uint32 x = blockIdx.x * blockDim.x + threadIdx.x;
    uint8* Buffer_ptr = buffer;
    SignalGroup_A_Type* Data_ptr = dataElement;

    if (x < NUM_BYTES) {
        uint32 idx = x;

        if (idx >= offsets1[0] && idx < offsets1[1]) {
            *((uint8*)(&(Data_ptr->signal1)) + (idx - offsets1[0])) = *(Buffer_ptr + idx);
        }
        if (idx >= offsets1[2] && idx < offsets1[3]) {
            *((uint8*)(&(Data_ptr->signal2)) + (idx - offsets1[2])) = *(Buffer_ptr + idx);
        }
        if (idx >= offsets1[4] && idx < offsets1[5]) {
            *((uint8*)(&(Data_ptr->signal3)) + (idx - offsets1[4])) = *(Buffer_ptr + idx);
        }
    }
    





}

uint8 ComXf_Inv_Com_ComSignalGroupA(uint8* buffer, uint32 bufferLength, SignalGroup_A_Type* dataElement) {

    uint32 block_size = 1024;
    uint32 grid_size = (NUM_BYTES + block_size - 1) / block_size;
    uint32 bufferlength = bufferLength;

    SignalGroup_A_Type* ptr_s = dataElement;
    SignalGroup_A_Type* dev_ptr_s = 0;



    /*auto start_time = std::chrono::high_resolution_clock::now();*/
    hipError_t cudaStatus = hipMalloc((void**)&dev_ptr_s, sizeof(SignalGroup_A_Type));
    if (cudaStatus != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
    }


     //Transfer data from host to device
    //cudaStatus = hipMemcpy(dev_ptr_s, ptr_s, sizeof(SignalGroup_A_Type), hipMemcpyHostToDevice);
    //if (cudaStatus != hipSuccess) {
    //    printf("hipMemcpy failed (HostToDevice): %s\n", hipGetErrorString(cudaStatus));
    //    hipFree(dev_ptr_s);  // Free the allocated device memory
    //}

    const uint32 offsets_size = 6;
    uint32 offsets_k1[offsets_size];
    uint32* d_offsets;

    uint32 start = 0, end = sizeof(dev_ptr_s->signal1);
    offsets_k1[0] = start;
    offsets_k1[1] = end;

    start = end;
    end = start + sizeof(dev_ptr_s->signal2);
    offsets_k1[2] = start;
    offsets_k1[3] = end;

    start = end;
    end = start + sizeof(dev_ptr_s->signal3);
    offsets_k1[4] = start;
    offsets_k1[5] = end;


   




    hipMalloc((void**)&d_offsets, offsets_size * sizeof(int));
    hipMemcpy(d_offsets, offsets_k1, offsets_size * sizeof(int), hipMemcpyHostToDevice);

    //auto start_time = std::chrono::high_resolution_clock::now();

    ComXf_Inv_Com_ComSignalGroupA_k << <grid_size, block_size >> > ((uint8*)buffer, bufferlength, dev_ptr_s, d_offsets);

    hipDeviceSynchronize();

    
    hipMemcpy(dataElement, dev_ptr_s, bufferlength, hipMemcpyDeviceToHost); //Transfer struct to host memory
  

 


    hipFree(dev_ptr_s);
    hipFree(d_offsets);

 

    return E_OK;

    

}
